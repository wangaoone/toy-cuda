
#include <hip/hip_runtime.h>
#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 10000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    int ha[N], hb[N];

    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    //for (int i = 0; i<N; ++i) {
    //    printf("%d\n", hb[i]);
    //}

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}
