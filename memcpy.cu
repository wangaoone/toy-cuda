
#include <hip/hip_runtime.h>
int main()
{
    const unsigned long long int X=1024 * 1024 * 512;
    const unsigned long long int bytes = X*sizeof(long long int);
    int *hostArray= (int*)malloc(bytes);
    int *deviceArray;
    hipMalloc((int **)&deviceArray, bytes);
    memset(hostArray, 10, bytes);
    hipMemcpy(deviceArray,hostArray,bytes,hipMemcpyHostToDevice);
    hipMemcpy(hostArray,deviceArray,bytes,hipMemcpyDeviceToHost);

    hipFree(deviceArray);

}
